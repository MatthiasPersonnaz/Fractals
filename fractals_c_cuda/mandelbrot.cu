#include "hip/hip_runtime.h"
// Compile with
// nvcc mandelbrot.cu -o mandelbrot -ccbin "C:\Program Files\Microsoft Visual Studio\2022\Community\VC\Tools\MSVC\14.33.31629\bin\Hostx64\x64" -O2

//#include <stdlib.h>
#include <stdio.h>
//#include <math.h>

#include <cuda/std/complex>

typedef unsigned char  byte;   // 0..255
typedef unsigned short ushort; // 0..65535
typedef unsigned int   uint;   // 0..4294967295

typedef struct HSVColor {
    float h;
    float s;
    float v;
} hsvcolor;

typedef struct RGBColor {
    byte r;
    byte g;
    byte b;
} rgbcolor;

// window dimensions and bounds
const int   WIDTH    = 4000;
const int   HEIGHT   = 3000;
const float LEFT_X   = -1.65;
const float RIGHT_X  = +1.65;
const float BOTTOM_Y = -1.2;
const float TOP_Y    =  1.2;

// maximum number of iterations
const ushort NB_ITER = 256;
const float C_REAL = -0.8;
const float C_IMAG = +0.156;

// output file
const char* filename = "mandelbrot_cu.bmp";

// bitmap file specifications
const int FILE_HEADER_SIZE = 14;
const int INFO_HEADER_SIZE = 40;
const int BYTES_PER_PIXEL  = 3;


unsigned char* createBitmapFileHeader(int height, int stride) {
    int fileSize = FILE_HEADER_SIZE + INFO_HEADER_SIZE + (stride * height);

    static unsigned char fileHeader[] = {
        0,0,     /// signature
        0,0,0,0, /// image file size in bytes
        0,0,0,0, /// reserved
        0,0,0,0, /// start of pixel array
    };

    fileHeader[ 0] = (unsigned char)('B');
    fileHeader[ 1] = (unsigned char)('M');
    fileHeader[ 2] = (unsigned char)(fileSize      );
    fileHeader[ 3] = (unsigned char)(fileSize >>  8);
    fileHeader[ 4] = (unsigned char)(fileSize >> 16);
    fileHeader[ 5] = (unsigned char)(fileSize >> 24);
    fileHeader[10] = (unsigned char)(FILE_HEADER_SIZE + INFO_HEADER_SIZE);

    return fileHeader;
}

unsigned char* createBitmapInfoHeader(int height, int width) {
    static unsigned char infoHeader[] = {
        0,0,0,0, /// header size
        0,0,0,0, /// image width
        0,0,0,0, /// image height
        0,0,     /// number of color planes
        0,0,     /// bits per pixel
        0,0,0,0, /// compression
        0,0,0,0, /// image size
        0,0,0,0, /// horizontal resolution
        0,0,0,0, /// vertical resolution
        0,0,0,0, /// colors in color table
        0,0,0,0, /// important color count
    };

    infoHeader[ 0] = (unsigned char)(INFO_HEADER_SIZE);
    infoHeader[ 4] = (unsigned char)(width      );
    infoHeader[ 5] = (unsigned char)(width >>  8);
    infoHeader[ 6] = (unsigned char)(width >> 16);
    infoHeader[ 7] = (unsigned char)(width >> 24);
    infoHeader[ 8] = (unsigned char)(height      );
    infoHeader[ 9] = (unsigned char)(height >>  8);
    infoHeader[10] = (unsigned char)(height >> 16);
    infoHeader[11] = (unsigned char)(height >> 24);
    infoHeader[12] = (unsigned char)(1);
    infoHeader[14] = (unsigned char)(BYTES_PER_PIXEL*8);

    return infoHeader;
}

void generateBitmapImage(unsigned char* image, int height, int width, const char* imageFileName) {
    int widthInBytes = width * BYTES_PER_PIXEL;

    unsigned char padding[3] = {0, 0, 0};
    int paddingSize = (4 - (widthInBytes) % 4) % 4;

    int stride = (widthInBytes) + paddingSize;

    FILE* imageFile = fopen(imageFileName, "wb");

    unsigned char* fileHeader = createBitmapFileHeader(height, stride);
    fwrite(fileHeader, 1, FILE_HEADER_SIZE, imageFile);

    unsigned char* infoHeader = createBitmapInfoHeader(height, width);
    fwrite(infoHeader, 1, INFO_HEADER_SIZE, imageFile);

    for (int i = 0; i < height; i++) {
        fwrite(image + (i*widthInBytes), BYTES_PER_PIXEL, width, imageFile);
        fwrite(padding, 1, paddingSize, imageFile);
    }

    fclose(imageFile);
}

rgbcolor hsv2rgb(hsvcolor c_in) {
    float h = c_in.h;
    float s = c_in.s;
    float v = c_in.v;
    float r, g, b;

    float i = floor(h * 6);
    float f = h * 6 - i;
    float p = v * (1 - s);
    float q = v * (1 - f * s);
    float t = v * (1 - (1 - f) * s);

    switch((int)i % 6){
        case 0: r = v, g = t, b = p; break;
        case 1: r = q, g = v, b = p; break;
        case 2: r = p, g = v, b = t; break;
        case 3: r = p, g = q, b = v; break;
        case 4: r = t, g = p, b = v; break;
        case 5: r = v, g = p, b = q; break;
    }

    rgbcolor c_out;
    c_out.r = (int)(r * 255);
    c_out.g = (int)(g * 255);
    c_out.b = (int)(b * 255);

    return c_out;
}

__global__ void mandel_iter(ushort *iter) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // handling arbitrary vector size
    if (tid < WIDTH * HEIGHT) {
        // compute coords
        float i = (float)(tid / WIDTH);
        float j = (float)(tid % WIDTH);
        
        // compute complex
        cuda::std::complex<float> z(LEFT_X + ((j / (float)WIDTH) * (RIGHT_X - LEFT_X)),
                                    TOP_Y - ((i / (float)HEIGHT) * (TOP_Y - BOTTOM_Y)));
        
        // execute algorithm
        cuda::std::complex<float> c(C_REAL, C_IMAG);
        ushort n = 0;
        while ((abs(z) < 2.) && (n < NB_ITER)) {
            z = z*z + c;
            n += 1;
        }
        iter[tid] = n;
    }
}

int offset(int i, int j, int color) {
    return WIDTH*BYTES_PER_PIXEL*i + BYTES_PER_PIXEL*j + color; 
}

int main(int argc, char **argv) {

    // arrays storing the number of iterations for each pixel
    ushort *h_iter;
    ushort *d_iter;
    size_t size_iter = WIDTH * HEIGHT * sizeof(ushort);
    
    // allocate host memory
    h_iter = (ushort*)malloc(size_iter);
    
    // allocate device memory
    hipMalloc((void**)&d_iter, size_iter);
    
    // transfer data from host memory to device memory
    hipMemcpy(d_iter, h_iter, size_iter, hipMemcpyHostToDevice);
    
    // executing kernel
    int block_size = 512;
    int grid_size = ((WIDTH*HEIGHT + block_size) / block_size);
    mandel_iter<<<grid_size, block_size>>>(d_iter);
    
    // transfer data back to host memory
    hipMemcpy(h_iter, d_iter, size_iter, hipMemcpyDeviceToHost);

    // generate RGB bitmap image
    byte* image = (byte*)malloc(HEIGHT * WIDTH * BYTES_PER_PIXEL * sizeof(byte));
    for (int i=0; i<HEIGHT; i++) {
        for (int j=0; j<WIDTH; j++) {
            int n_iter = h_iter[i*WIDTH + j];
            //printf("%d\n", n_iter);
            
            hsvcolor hsv;
            hsv.h = (float)n_iter / (float)NB_ITER;
            hsv.s = 1.0;
            if (n_iter < NB_ITER) { hsv.v = 1.0; } else { hsv.v = 0.0; }
            rgbcolor rgb = hsv2rgb(hsv);
            
            image[offset(i, j, 0)] = (byte)(rgb.r); // red
            image[offset(i, j, 1)] = (byte)(rgb.g); // green
            image[offset(i, j, 2)] = (byte)(rgb.b); // blue
        }
    }
    generateBitmapImage((byte*)image, HEIGHT, WIDTH, filename);
    free(image);
    
    // deallocate device memory
    hipFree(d_iter);

    // deallocate host memory
    free(h_iter);

    return 0;
}
